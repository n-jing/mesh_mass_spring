#include "mesh.h"
#include "remove_duplicate_vert.h"
#include "solver.h"
#include "writer.h"
#include "time_integral.h"
#include <iostream>
#include <igl/readOBJ.h>
#include <igl/writeOBJ.h>
#include <random>
#include <Eigen/Core>
#include <string>


using namespace std;
using namespace Eigen;
using namespace igl;

int main (int argc, char *argv[])
{
  remove_duplicate_vert(argv[1], "mesh.obj");
  MatrixXd V;
  MatrixXi F;
  readOBJ("mesh.obj", V, F);


  double *v;
  int *f;
  EdgeMesh *edge_mesh;
  hipMallocManaged((void**)&edge_mesh, sizeof(EdgeMesh));
  hipMallocManaged((void**)&v, V.size() * sizeof(double));
  hipMallocManaged((void**)&f, F.size() * sizeof(int));
  hipMallocManaged((void**)&edge_mesh->vert_, V.rows() * sizeof(EdgeMesh::Vert));
  hipMallocManaged((void**)&edge_mesh->edge_, 3*F.rows()*sizeof(EdgeMesh::Edge));

  for (int i = 0; i < V.size(); ++i)
    v[i] = V[i];
  for (int i = 0; i < F.size(); ++i)
    f[i] = F[i];

  edge_mesh->init(v, V.rows(), V.cols(), f, F.rows(), F.cols());
  cerr << edge_mesh->get_vert_num() << " " << edge_mesh->get_edge_num() << endl;

  const int vert_num = edge_mesh->get_vert_num();
  const int var_num = 3 * (vert_num - 1);
  const double time = 4;
  const double delta_t = 1e-4;
  
  random_device rd;
  mt19937 gen(rd());
  uniform_int_distribution<int> vert_u(0, vert_num-1);

  edge_mesh->fixed_vert = vert_u(gen);

  double *var;
  double *speed;
  double *vert;
  hipMallocManaged((void**)&var, var_num * sizeof(double));
  hipMallocManaged((void**)&speed, var_num * sizeof(double));
  hipMallocManaged((void**)&vert, 3*vert_num * sizeof(double));

  dim3 block_size(256);
  dim3 grid_size((edge_mesh->get_vert_num() + block_size.x - 1) / block_size.x);
  init_var_and_speed<<<grid_size, block_size>>>(var, speed, edge_mesh);

  get_vert(var, vert, edge_mesh);
  hipDeviceSynchronize();
  write_mesh_to_vtk(&init_edge_vert[0], edge_mesh, "init_state.vtk");
  
  Integral integral = Integral::location_implicit;
  time_integral(var, speed, time, delta_t, integral, edge_mesh, vert);
  
  get_vert(var, vert, edge_mesh);
  hipDeviceSynchronize();
  write_mesh_to_vtk(&balance_edge_vert[0], edge_mesh, "final_state.vtk");

  hipFree(v);
  hipFree(f);
  hipFree(edge_mesh->vert_);
  hipFree(edge_mesh->edge_);
  hipFree(var);
  hipFree(speed);
  hipFree(vert);
  return 0;
}
